#include "hip/hip_runtime.h"
#include "../knn.h"
#include "hipcub/hipcub.hpp"
#include <limits>

namespace knn
{
#include "kernel.cuh"

// device data
template <typename T>
struct Device
{
  T *pts;
  T *dimDelt2s; // (pt_i_dim_d - pt_j_dim_d)**2
  T *dimDelt2Sums; //sum of dimDelt2s across the ndim dimensions
  T *dists;
  int* indexes;
  void *tmp; //used by hipcub::min
  hipcub::KeyValuePair<int, T> *indexDist;
  T *dist;
};
// host data
struct Host
{
  int k; //number of nearest neighbors
  int ndim; //number of dimensions
  int npt; //number of points
  size_t cubTmpSize; //bytes of storage required by hipcub::min
  bool findIndexes;
  bool findDistances;
};

template <typename T>
void allocate_device_storage( Host *h, Device<T> *d )
{
  size_t bytes = sizeof(T) * h->npt * h->ndim;
  hipError_t err = hipMalloc( &d->pts, bytes );
  bytes = sizeof(T) * h->npt;
  err = hipMalloc( &d->dimDelt2s, bytes );
  err = hipMalloc( &d->dimDelt2Sums, bytes );
  d->tmp = NULL;
  if (h->findDistances && !h->findIndexes)
  {
    bytes = sizeof(T) * h->npt * h->k;
    err = hipMalloc( &d->dists, bytes );
      bytes = sizeof(T);
      err = hipMalloc( &d->dist, bytes );
      hipcub::DeviceReduce::Min( 
        d->tmp, 
        h->cubTmpSize, 
        d->dists, 
        d->dist,
        h->npt );
  }
  else
  {
    bytes = sizeof(int) * h->npt * h->k;
    err = hipMalloc( &d->indexes, bytes);
    bytes = sizeof(hipcub::KeyValuePair<int, T>);
    err = hipMalloc( &d->indexDist, bytes );
    hipcub::DeviceReduce::ArgMin( 
      d->tmp, 
      h->cubTmpSize, 
      d->dists, 
      d->indexDist,
      h->npt );
  }
  return;
}

template <typename T>
void free_device_storage( Device<T> *d )
{
  if (d->pts != NULL) hipFree( d->pts );
  if (d->dimDelt2s != NULL) hipFree( d->dimDelt2s );
  if (d->dimDelt2Sums != NULL) hipFree( d->dimDelt2Sums );
  if (d->dists != NULL) hipFree( d->dists );
  if (d->indexes != NULL) hipFree( d->indexes );
  if (d->tmp != NULL) hipFree( d->tmp );
  if (d->indexDist != NULL) hipFree( d->indexDist );
  if (d->dist != NULL) hipFree( d->dist );
  return;
}

template <typename T>
void add_to_dimDelt2Sums( Host *h, Device<T> *d, const int ipt, const int idim )
{
  dim3 bsize (BLOCK_DIMX,1,1);
  dim3 gsize (h->npt/bsize.x,1,1);
  compute_dimDelt2s_kernel<<<gsize,bsize>>>(
    d->dimDelt2s, 
    d->dimDelt2Sums, 
    ipt, 
    idim );
  return;
}

template <typename T>
void find_ith_neighbor( Host *h, Device<T> *d, const int ipt, const int inn )
{
  if (h->findDistances  && !h->findIndexes)
  {
    hipcub::DeviceReduce::Min( 
      d->tmp, 
      h->cubTmpSize, 
      d->dists, 
      d->dist,
      h->npt );
  }
  else
  {
    hipcub::DeviceReduce::ArgMin( 
      d->tmp, 
      h->cubTmpSize, 
      d->dists, 
      d->indexDist,
      h->npt );
  }

  return;
}

void zero_dimDelt2Sums()
{
  return;
}

void set_dist_to_max( const int ipt )
{
  return;
}

} /*namespace knn*/

void knn_indexes( 
  const int k, 
  const int ndim, 
  const int npt, 
  const float *const pts_in, 
  int *const indexes_out )
{
  knn::Host hostParams;
  hostParams.k = k;
  hostParams.ndim = ndim;
  hostParams.npt = npt;
  hostParams.cubTmpSize = 0;
  hostParams.findIndexes = true;
  hostParams.findDistances = false;
  knn::Device<float> deviceParams;
  knn::allocate_device_storage<float>( &hostParams, &deviceParams);
  knn::free_device_storage<float>( &deviceParams );
  return;
}

